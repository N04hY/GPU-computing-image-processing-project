#include "hip/hip_runtime.h"
#include "Func.h"

// tiled approach
// #매크로
__global__ void Kern_Grayscale(uint8_t* buf, uint8_t* gray, uint8_t start_add, int len) {
	int i = threadIdx.x;

	if (i >= start_add && i < len && (i % 3) == (start_add % 3)) {
		int tmp = (buf[i] * 0.114 + buf[i + 1] * 0.587 + buf[i + 2] * 0.299);
		gray[i] = tmp;
		gray[i + 1] = tmp;
		gray[i + 2] = tmp;
	}
}

void GPU_Grayscale(uint8_t* buf, uint8_t* gray, uint8_t start_add, int len) {
	printf("%d\n",start_add);
	uint8_t* g_buf;
	uint8_t* g_gray;

	hipMalloc(&g_buf, len);
	hipMalloc(&g_gray, len);

	hipMemcpy(g_buf, buf, len, hipMemcpyHostToDevice);
	hipMemcpy(g_gray, gray, len, hipMemcpyHostToDevice);
/**/
	cout << len;
	Kern_Grayscale <<<1, len>>> (g_buf, g_gray, start_add, len);

	hipMemcpy(buf, g_buf, len, hipMemcpyDeviceToHost);
	hipMemcpy(gray, g_gray, len, hipMemcpyDeviceToHost);

	hipFree(g_gray);
	hipFree(g_buf);
}


float conv2d_5x5(float* filter, uint8_t* pixel, int x, int y, int width) {
	float v = 0;
	for (int i = 0; i < 5; i++) {
		for (int j = 0; j < 5; j++) {
			v += pixel[(y + i) * width + x + j] * filter[i * 5 + j];
		}
	}
	return v;
}

void GPU_Noise_Reduction(int width, int height, uint8_t *gray, uint8_t *gaussian) {
// noise blurring
// gaussian blur 5x5 filter
// 2d convolution
	float filter[25] = {0}; 
	float sigma = 1.0;
	for (int i = -2; i <= 2; i++) {
		for (int j = -2; j <= 2; j++) {
			filter[(i + 2) * 5 + j + 2]
				= (1 / (2 * 3.14* sigma * sigma)) * exp(-(i * i + j * j) / (2 * sigma * sigma));
		}
	}

	//zero padding
	uint8_t* tmp = (uint8_t*)malloc((width+4) * (height+4));
	memset(tmp, (uint8_t)0, (width + 4) * (height + 4));


	for (int i = 2; i < height+2; i++) {
		for (int j = 2; j < width+2; j++) {
			tmp[i * (width + 4) + j] = gray[((i - 2) * width + (j - 2)) * 3];
		}
	}
	
	//GaussianBlur

	for (int i = 0; i < height; i++) {
		for (int j = 0; j < width; j++) {
			uint8_t v = conv2d_5x5(filter,tmp,j, i,width+4);
			gaussian[(i * width + j)*3] = v;
			gaussian[(i * width + j) * 3 +1] = v;
			gaussian[(i * width + j) * 3 +2] = v;
		}
	}
	free(tmp);
}

void conv2d_3x3(int* filter_y, int* filter_x, uint8_t* pixel, int x, int y, int width, int &gx, int &gy) {
	//int gx = 0;
	//int gy = 0;
	for (int i = 0; i < 3; i++) {
		for (int j = 0; j < 3; j++) {
			gy += (int)pixel[(y + i) * width + x + j] * filter_y[i *3 + j];
			gx += (int)pixel[(y + i) * width + x + j] * filter_x[i * 3 + j];
		}
		// image의 pixel이 edge인지 아닌지 검출하기 위하여 인접 pixel과의 gradient를 구하고 
	}
	// -> 각 픽셀의 x,y축 방향의 gradient를 연산하고 equation2를 통해 gradient값과 방향구함
}
,-2,0,2

int filter_y[9] = {1,2,1

	,-1,-2,-1};
void GPU_Intensity_Gradient(int width, int height, uint8_t* gaussian, uint8_t* sobel, uint8_t*angle){
		uint8_t* tmp = (uint8_t*)malloc((width + 2) * (height + 2));
	// 어떤 방향의 gradient인지 구하기 -> sobel filter와 image를 2D convolution연산
			
	int filter_x[9] = {-1,0,1
		
		,-1,0,1};
		for (int j = 1; j < width + 1; j++) {
		,0,0,0
		}
		
		for (int i = 0; i < height; i++) {
	memset(tmp, (uint8_t)0, (width + 2) * (height + 2));
		int gx = 0;
	//zero padding
		conv2d_3x3(filter_y, filter_x, tmp, j, i, width + 2,gx,gy);
	for (int i = 1; i < height + 1; i++) {
		uint8_t  v = 0;
	tmp[i * (width + 2) + j] = gaussian[((i - 1) * width + (j - 1)) * 3];
	v = 255;
	}
	else
	for (int j = 0; j < width; j++) {
	
	int gy = 0;
	sobel[(i * width + j) * 3 + 1] = v;
	int t = sqrt(gx * gx + gy * gy);
		
	if (t > 255) {
	if(gy != 0 || gx != 0) 
	}
	if ((t_angle > -22.5 && t_angle <= 22.5) || (t_angle > 157.5 || t_angle <= -157.5))
	v = t;
	else if ((t_angle > 22.5 && t_angle <= 67.5) || (t_angle > -157.5 && t_angle <= -112.5))
	sobel[(i * width + j) * 3] = v;
	else if ((t_angle > 67.5 && t_angle <= 112.5) || (t_angle > -112.5 && t_angle <= -67.5))
	sobel[(i * width + j) * 3 + 2] = v;
	else if ((t_angle > 112.5 && t_angle <= 157.5) || (t_angle > -67.5 && t_angle <= -22.5))
	float t_angle = 0;
}
	t_angle= (float)atan2(gy, gx) * 180.0 / 3.14;
	free(tmp);
	angle[i * width + j] = 0;
	angle[i * width + j] = 45;
	angle[i * width + j] = 90;
	angle[i * width + j] = 135;
	}

}


void GPU_Non_maximum_Suppression(int width, int height, uint8_t *angle,uint8_t *sobel, uint8_t *suppression_pixel, uint8_t& min, uint8_t& max){
	uint8_t p1 = 0;
	uint8_t p2 = 0;
	for (int i = 1; i < height-1; i++) {
		for (int j = 1; j < width-1; j++) {
			if (angle[i * width + j] == 0) {
				p1 = sobel[((i+1) * width + j)*3];
				p2 = sobel[((i-1) * width + j) * 3];
			}
			else if (angle[i * width + j] == 45) {
				p1 = sobel[((i + 1) * width + j-1) * 3];
				p2 = sobel[((i - 1) * width + j+1) * 3];
			}
			else if (angle[i * width + j] == 90) {
				p1 = sobel[((i) * width + j+1) * 3];
				p2 = sobel[((i) * width + j-1) * 3];
			}
			else {
				p1 = sobel[((i + 1) * width + j+1) * 3];
				p2 = sobel[((i - 1) * width + j-1) * 3];
			}
			uint8_t v = sobel[(i * width + j) * 3];
			if(min > v)
				min = v;
			if(max < v)
				max = v;
			if ((v >= p1) && (v >= p2)) {
				suppression_pixel[(i * width + j) * 3] = v;
				suppression_pixel[(i * width + j) * 3 + 1] = v;
				suppression_pixel[(i * width + j) * 3 + 2] = v;
			}
			else {
				suppression_pixel[(i * width + j) * 3] = 0;
				suppression_pixel[(i * width + j) * 3 + 1] = 0;
				suppression_pixel[(i * width + j) * 3 + 2] = 0;
			}
		}
	}


}


void Hysteresis_check(int width, int height, int x, int y, uint8_t * hysteresis, uint8_t *tmp_hysteresis){
	for (int i = y-1; i < y+2; i++) {
		for (int j = x-1; j < x+2; j++) {
			if ((i < height && j < width) && (i >= 0 && j >= 0)) {
				if (tmp_hysteresis[(i * width + j)*3] == 255) {
					hysteresis[(y * width + x)*3] = 255;
					hysteresis[(y * width + x) * 3+1] = 255;
					hysteresis[(y * width + x) * 3+2] = 255;
					return;
				}
			}
		}
	}
}


void GPU_Hysteresis_Thresholding(int width, int height, uint8_t *suppression_pixel,uint8_t *hysteresis, uint8_t min, uint8_t max) {
	uint8_t diff = max - min;
	uint8_t low_t = min + diff * 0.01;
	uint8_t high_t = min + diff * 0.2;
	uint8_t *tmp_hysteresis = (uint8_t*)malloc(sizeof(uint8_t)*width*height*3);

	for (int i = 0; i < height; i++) {
		for (int j = 0; j < width; j++) {
			uint8_t v = suppression_pixel[(i * width + j)*3];
			if (v < low_t) {
				hysteresis[(i * width + j) * 3] = 0;
				hysteresis[(i * width + j) * 3+1] = 0;
				hysteresis[(i * width + j) * 3+2] = 0;
			}
			else if (v < high_t) {
				hysteresis[(i * width + j) * 3] = 123;
				hysteresis[(i * width + j) * 3 + 1] = 123;
				hysteresis[(i * width + j) * 3 + 2] = 123;
			}
			else {
				hysteresis[(i * width + j) * 3] = 255;
				hysteresis[(i * width + j) * 3 + 1] = 255;
				hysteresis[(i * width + j) * 3 + 2] = 255;
			}
		}
	}
//////////////////////Modified in Version3//////////////////////////////
	memcpy(tmp_hysteresis,hysteresis,sizeof(uint8_t)*width*height*3);
    	for (int i = 0; i < height; i++) {
		for (int j = 0; j < width; j++) {
			if(tmp_hysteresis[(i*width+j)*3] == 123){
                		Hysteresis_check(width,height,j,i,hysteresis,tmp_hysteresis);
			}
		}
	}
//////////////////////////////////////////////////////////////////////
	for (int i = 0; i < height; i++) {
		for (int j = 0; j < width; j++) {
			if (hysteresis[(i * width + j) * 3] != 255) {
				hysteresis[(i * width + j) * 3] = 0;
				hysteresis[(i * width + j) * 3+1] = 0;
				hysteresis[(i * width + j) * 3+2] = 0;
			}
		}
	}
	free(tmp_hysteresis);

}

bool Image_Check(uint8_t *cpu, uint8_t *gpu, int len){
    for(int i=0;i<len;i++){
        if(cpu[i] != gpu[i] && cpu[i] + 1 != gpu[i] && cpu[i] - 1 != gpu[i]){ 
            return false;
        }
    }
    return true;
}
